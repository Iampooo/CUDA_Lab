// Lab 0: VS Setup and Hello World!

#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void kernel() {
	printf("Hello, World!\n");
}

int main() {
	kernel<<<1, 1>>>();
	hipDeviceSynchronize();
	return 0;
}